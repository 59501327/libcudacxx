#include "hip/hip_runtime.h"
#include <cassert>



#include <gpu/cstdint>
#include <gpu/cstddef>
#include <gpu/atomic>

// preview on GitHub, monthly release at head
//#include <gpu/mutex>

namespace cuda { namespace std {
    struct mutex { 
        __host__ __device__ bool try_lock() { return true; } 
        __host__ __device__ void lock() { } 
        __host__ __device__ void unlock() { } 
    };
}}

// stay tuned for <algorithm>
template<class T> static constexpr T min(T a, T b) { return a < b ? a : b; }

struct node {
    struct ref {
        cuda::std::atomic<node*> ptr = ATOMIC_VAR_INIT(nullptr);
        cuda::std::mutex         lock;
    };
    ref                    next[26];
    cuda::std::atomic<int> count = ATOMIC_VAR_INIT(0);
};
struct trie {
    cuda::std::atomic<node*> bump = ATOMIC_VAR_INIT(nullptr);
    node                     root;
    __host__ __device__ trie(node* ptr) : bump(ptr) { }
};

__host__ __device__ void process(const char* begin, const char* end, trie* t, unsigned const index, unsigned const range) {

    auto const size = end - begin;
    auto const stride = (size / range + 1);

    auto off = min(size, stride * index);
    auto const last = min(size, off + stride);

    auto const index_of = [](char c) -> int {
        if(c >= 'a' && c <= 'z') return c - 'a';
        if(c >= 'A' && c <= 'Z') return c - 'A';
        return -1;
    };

    for(char c = begin[off]; off < size && off != last && c != 0 && index_of(c) != -1; ++off, c = begin[off]);
    for(char c = begin[off]; off < size && off != last && c != 0 && index_of(c) == -1; ++off, c = begin[off]);

    node *const proot = &t->root, *n = proot;
    for(char c = begin[off]; ; ++off, c = begin[off]) {
        auto const index = off >= size ? -1 : index_of(c);
        if(index == -1) {
            if(n != proot) {
                n->count.fetch_add(1, cuda::std::memory_order_relaxed);
                n = proot;
            }
            //end of last word?
            if(off >= size || off > last)
                break;
            else
                continue;
        }
        auto& ptr = n->next[index].ptr;
        auto next = ptr.load(cuda::std::memory_order_acquire);
        if(next == nullptr) {
            auto& lock = n->next[index].lock;
            if(!lock.try_lock()) {
                do {
                    next = ptr.load(cuda::std::memory_order_acquire);
                } while(next == nullptr);
            }
            else {
                next = ptr.load(cuda::std::memory_order_acquire);
                if(next == nullptr) {
                    next = t->bump.fetch_add(1, cuda::std::memory_order_relaxed);
                    ptr.store(next, cuda::std::memory_order_relaxed);
                    lock.unlock();
	        }
	        else lock.unlock();
            }
        }
        n = next;
    }
}

__global__ __launch_bounds__(1024, 2) void call_process(const char* begin, const char* end, trie* t) {
    auto const index = blockDim.x * blockIdx.x + threadIdx.x;
    auto const range = gridDim.x * blockDim.x;
    process(begin, end, t, index, range);
}

#include <iostream>
#include <fstream>
#include <string>
#include <utility>
#include <vector>
#include <chrono>
#include <thread>

template <class T>
struct managed_allocator {
  typedef T value_type;
  managed_allocator() = default;
  template <class U> constexpr managed_allocator(const managed_allocator<U>&) noexcept {}
  [[nodiscard]] T* allocate(std::size_t n) {
    assert(n <= std::size_t(-1) / sizeof(T));
    void* out = nullptr;
    auto const ret = hipMallocManaged(&out, n*sizeof(T));
    assert(ret == hipSuccess);
    if(auto p = static_cast<T*>(out)) return p;
    return nullptr;
  }
  void deallocate(T* p, std::size_t) noexcept { 
      hipFree(p); 
  }
};
template<class T, class... Args>
T* make_(Args &&... args) {
    managed_allocator<T> ma;
    return new (ma.allocate(1)) T(std::forward<Args>(args)...);
}

using string = std::basic_string<char, std::char_traits<char>, managed_allocator<char>>;
using vector = std::vector<node, managed_allocator<node>>;

void do_trie(string* input, vector* nodes, bool use_gpu, int blocks, int threads) {
    
    hipMemset(nodes->data(), 0, nodes->size() * sizeof(node));

    trie* const t = make_<trie>(nodes->data());

    auto const begin = std::chrono::steady_clock::now();
    std::atomic_signal_fence(std::memory_order_seq_cst);
    if(use_gpu) {
        call_process<<<blocks,threads>>>(input->data(), input->data() + input->size(), t);
        auto const ret = hipDeviceSynchronize();
	assert(ret == hipSuccess);
    }
    else {
        assert(blocks == 1);
        std::vector<std::thread> tv(threads);
        for(auto count = threads; count; --count)
            tv[count - 1] = std::thread([&, count]() {
                process(input->data(), input->data() + input->size(), t, count - 1, threads);
            });
        for(auto& t : tv)
            t.join();
    }
    std::atomic_signal_fence(std::memory_order_seq_cst);
    auto const end = std::chrono::steady_clock::now();
    auto const time = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    auto const count = t->bump - nodes->data();
    std::cout << "Assembled " << count << " nodes on " << blocks << "x" << threads << " " << (use_gpu ? "gpu" : "cpu") << " threads in " << time << "ms." << std::endl;
}

int main() {

    string* input = make_<string>();
    vector* nodes = make_<vector>(1<<20);

    char const* files[10] = { "divine_comedy.txt",
                              "frankenstein.txt",
                              "iliad.txt",  
                              "moby_dick.txt",  
                              "odyssey.txt",  
                              "oz.txt",  
                              "quixote.txt",  
                              "time_machine.txt",
                              "war_and_peace.txt",
                              "quixote.txt" };

    std::size_t total = 0, cur = 0;
    for(auto* ptr : files) {
        std::ifstream in(ptr);
        in.seekg(0, std::ios_base::end);
        total += in.tellg();
    }
    input->resize(total);
    for(auto* ptr : files) {
        std::ifstream in(ptr);
        in.seekg(0, std::ios_base::end);
        auto const pos = in.tellg();
        in.seekg(0, std::ios_base::beg);
        in.read((char*)input->data() + cur, pos);
        cur += pos;
    }

    do_trie(input, nodes, false, 1, 1);
    do_trie(input, nodes, false, 1, 1);
    do_trie(input, nodes, false, 1, std::thread::hardware_concurrency());
    do_trie(input, nodes, false, 1, std::thread::hardware_concurrency());

    if(hipSetDevice(0) != hipSuccess) return 1;
    hipDeviceProp_t deviceProp;
    if(hipGetDeviceProperties(&deviceProp, 0) != hipSuccess) return 2;

    do_trie(input, nodes, true, deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor >> 10, 1<<10);
    do_trie(input, nodes, true, deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor >> 10, 1<<10);

    return 0;
}

