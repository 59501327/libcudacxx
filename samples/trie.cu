#include "hip/hip_runtime.h"
/*

Copyright (c) 2018, NVIDIA Corporation
All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
this list of conditions and the following disclaimer in the documentation
and/or other materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED
OF THE POSSIBILITY OF SUCH DAMAGE.

*/

#include <gpu/cstddef>
#include <gpu/cstdint>
#include <gpu/atomic>

#include "mutex.hpp"

// stay tuned for <algorithm>
template<class T> static constexpr T min(T a, T b) { return a < b ? a : b; }

struct node {
    struct ref {
        gpu::std::atomic<node*>  ptr = ATOMIC_VAR_INIT(nullptr);
        gpu::experimental::mutex lock;
    };
    ref                    next[26];
    gpu::std::atomic<int> count = ATOMIC_VAR_INIT(0);
};
struct trie {
    gpu::std::atomic<node*> bump = ATOMIC_VAR_INIT(nullptr);
    node                     root;
    __host__ __device__ trie(node* ptr) : bump(ptr) { }
};

__host__ __device__ void process(const char* begin, const char* end, trie* t, unsigned const index, unsigned const range) {

    auto const size = end - begin;
    auto const stride = (size / range + 1);

    auto off = min(size, stride * index);
    auto const last = min(size, off + stride);

    auto const index_of = [](char c) -> int {
        if(c >= 'a' && c <= 'z') return c - 'a';
        if(c >= 'A' && c <= 'Z') return c - 'A';
        return -1;
    };

    for(char c = begin[off]; off < size && off != last && c != 0 && index_of(c) != -1; ++off, c = begin[off]);
    for(char c = begin[off]; off < size && off != last && c != 0 && index_of(c) == -1; ++off, c = begin[off]);

    node *const proot = &t->root, *n = proot;
    for(char c = begin[off]; ; ++off, c = begin[off]) {
        auto const index = off >= size ? -1 : index_of(c);
        if(index == -1) {
            if(n != proot) {
                n->count.fetch_add(1, gpu::std::memory_order_relaxed);
                n = proot;
            }
            //end of last word?
            if(off >= size || off > last)
                break;
            else
                continue;
        }
        auto& ptr = n->next[index].ptr;
        auto next = ptr.load(gpu::std::memory_order_acquire);
        if(next == nullptr) {
            auto& lock = n->next[index].lock;
            if(!lock.try_lock()) {
                do {
                    next = ptr.load(gpu::std::memory_order_acquire);
                } while(next == nullptr);
            }
            else {
                next = ptr.load(gpu::std::memory_order_acquire);
                if(next == nullptr) {
                    next = t->bump.fetch_add(1, gpu::std::memory_order_relaxed);
                    ptr.store(next, gpu::std::memory_order_relaxed);
                    lock.unlock();
	        }
	        else lock.unlock();
            }
        }
        n = next;
    }
}

#ifdef __HIPCC__
__global__ __launch_bounds__(1024, 2) 
#endif
void call_process(const char* begin, const char* end, trie* t) {
    auto const index = blockDim.x * blockIdx.x + threadIdx.x;
    auto const range = gridDim.x * blockDim.x;
    process(begin, end, t, index, range);
}

#include <iostream>
#include <fstream>
#include <string>
#include <utility>
#include <vector>
#include <chrono>
#include <thread>
#include <atomic>
#include <cassert>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <class T>
struct managed_allocator {
  typedef T value_type;
  managed_allocator() = default;
  template <class U> constexpr managed_allocator(const managed_allocator<U>&) noexcept {}
  T* allocate(std::size_t n) {
    assert(n <= std::size_t(-1) / sizeof(T));
    void* out = nullptr;
    gpuErrchk(hipMallocManaged(&out, n*sizeof(T)));
    if(auto p = static_cast<T*>(out)) return p;
    return nullptr;
  }
  void deallocate(T* p, std::size_t) noexcept { 
      gpuErrchk(hipFree(p)); 
  }
};
template<class T, class... Args>
T* make_(Args &&... args) {
    managed_allocator<T> ma;
    return new (ma.allocate(1)) T(std::forward<Args>(args)...);
}

using string = std::basic_string<char, std::char_traits<char>, managed_allocator<char>>;
using vector = std::vector<node, managed_allocator<node>>;

void do_trie(string* input, vector* nodes, bool use_gpu, int blocks, int threads) {
    
    gpuErrchk(hipMemset(nodes->data(), 0, nodes->size() * sizeof(node)));

    trie* const t = make_<trie>(nodes->data());

    auto const begin = std::chrono::steady_clock::now();
    std::atomic_signal_fence(std::memory_order_seq_cst);
    if(use_gpu) {
        call_process<<<blocks,threads>>>(input->data(), input->data() + input->size(), t);
        gpuErrchk(hipGetLastError());
        gpuErrchk(hipDeviceSynchronize());
    }
    else {
        assert(blocks == 1);
        std::vector<std::thread> tv(threads);
        for(auto count = threads; count; --count)
            tv[count - 1] = std::thread([&, count]() {
                process(input->data(), input->data() + input->size(), t, count - 1, threads);
            });
        for(auto& t : tv)
            t.join();
    }
    std::atomic_signal_fence(std::memory_order_seq_cst);
    auto const end = std::chrono::steady_clock::now();
    auto const time = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
    auto const count = t->bump - nodes->data();
    std::cout << "Assembled " << count << " nodes on " << blocks << "x" << threads << " " << (use_gpu ? "gpu" : "cpu") << " threads in " << time << "ms." << std::endl;
}

int main() {

    string* input = make_<string>();
    vector* nodes = make_<vector>(1<<20);

    char const* files[] = {
        "2600-0.txt", "2701-0.txt", "35-0.txt", "84-0.txt", "8800.txt",
      	"pg1727.txt", "pg55.txt", "pg6130.txt", "pg996.txt", "1342-0.txt"
    };

    std::size_t total = 0, cur = 0;
    for(auto* ptr : files) {
        std::ifstream in(ptr);
        in.seekg(0, std::ios_base::end);
        total += in.tellg();
    }
    input->resize(total);
    for(auto* ptr : files) {
        std::ifstream in(ptr);
        in.seekg(0, std::ios_base::end);
        auto const pos = in.tellg();
        in.seekg(0, std::ios_base::beg);
        in.read((char*)input->data() + cur, pos);
        cur += pos;
    }

    do_trie(input, nodes, false, 1, 1);
    do_trie(input, nodes, false, 1, 1);
    do_trie(input, nodes, false, 1, std::thread::hardware_concurrency());
    do_trie(input, nodes, false, 1, std::thread::hardware_concurrency());

    gpuErrchk(hipSetDevice(0));
    hipDeviceProp_t deviceProp;
    gpuErrchk(hipGetDeviceProperties(&deviceProp, 0));

    do_trie(input, nodes, true, deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor >> 10, 1<<10);
    do_trie(input, nodes, true, deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor >> 10, 1<<10);

    return 0;
}

